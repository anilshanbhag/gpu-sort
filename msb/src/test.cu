// nvcc test.cu sort/*.cu ../external/benchmark/*.cu -O3 -arch=sm_52 -I. -I../external -lcurand

#include "sort/gpu_radix_sort.h"
#include "hipcub/hipcub.hpp"
#include <hiprand.h>

using namespace std;
using namespace hipcub;

#define SETUP_TIMING() hipEvent_t start, stop; hipEventCreate(&start); hipEventCreate(&stop);

#define TIME_FUNC(f,t) { \
    hipEventRecord(start, 0); \
    f; \
    hipEventRecord(stop, 0); \
    hipEventSynchronize(stop); \
    hipEventElapsedTime(&t, start,stop); \
}


void test_sort_keys(unsigned int num_keys)
{
  uint *d_key_buf;
  uint *d_val_buf;
  uint *d_key_backup;
  uint *d_val_backup;
  uint *d_key_alt_buf;
  uint *d_val_alt_buf;

  CubDebugExit(hipMalloc((void**)&d_key_buf, sizeof(float) * num_keys));
  CubDebugExit(hipMalloc((void**)&d_key_backup, sizeof(float) * num_keys));
  CubDebugExit(hipMalloc((void**)&d_key_alt_buf, sizeof(float) * num_keys));
  CubDebugExit(hipMalloc((void**)&d_val_buf, sizeof(float) * num_keys));
  CubDebugExit(hipMalloc((void**)&d_val_backup, sizeof(float) * num_keys));
  CubDebugExit(hipMalloc((void**)&d_val_alt_buf, sizeof(float) * num_keys));

  hiprandGenerator_t generator;
  int seed = 0;

  hiprandCreateGenerator(&generator, HIPRAND_RNG_PSEUDO_DEFAULT);
  hiprandSetPseudoRandomGeneratorSeed(generator,seed);
  hiprandGenerate(generator, d_key_buf, num_keys);
  hiprandGenerate(generator, d_val_buf, num_keys);

  hipMemcpy(d_key_backup, d_key_buf, sizeof(uint) * num_keys, hipMemcpyDeviceToDevice);
  hipMemcpy(d_val_backup, d_val_buf, sizeof(uint) * num_keys, hipMemcpyDeviceToDevice);

  SETUP_TIMING();
  int num_trials = 1;
  for (int i=0; i<num_trials; i++) {
    float time_sort_k;
    float time_sort_kv;
    TIME_FUNC((rdxsrt_unstable_sort<uint, hipcub::NullType, unsigned int>(d_key_buf, NULL, num_keys, d_key_alt_buf, NULL)), time_sort_k);
    cout << "Time Sort K: " << time_sort_k << endl;
    TIME_FUNC((rdxsrt_unstable_sort<uint, uint, unsigned int>(d_key_buf, d_val_buf, num_keys, d_key_alt_buf, d_val_alt_buf, NULL)), time_sort_kv);
    cout << "Time Sort KV: " << time_sort_kv << endl;

    hipMemcpy(d_key_buf, d_key_backup, sizeof(uint) * num_keys, hipMemcpyDeviceToDevice);
    hipMemcpy(d_val_buf, d_val_backup, sizeof(uint) * num_keys, hipMemcpyDeviceToDevice);
  }
}

int main() {
  test_sort_keys(1 << 28);
  return 0;
}
